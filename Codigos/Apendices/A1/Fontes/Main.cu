#include <iostream>
#include <string>

#include "Fontes/Macros/MacrosSO.h"
#include "Fontes/MonteCarlo.h"

using std::cout;
using std::endl;
using std::string;
using std::to_string;

void help(string exe) {
  cout << "Uso: " << exe << " <opcoes>\n"
       << "Opcoes:\n"
       << "\t-h\t\tMostra esta ajuda\n"
       << "\t-d ID_DEVICE\tEspecifica a GPU para execucao\n"
       << "\t-m QUANT_MCS\tEspecifica a quantidade de simulacoes MC\n";
}

int main(int argc, char **argv) {
  int idDevice = 0, quantMCs = 1;

  if (argc > 1) {
    for (int i = 1; i < argc; i += 2) {
      string textoOpcao(argv[i]);

      if (textoOpcao == "-h") {
        help(argv[0]);
        return 0;
      }
      if (textoOpcao == "-d") {
        idDevice = atoi(argv[i + 1]);
      } 
      if (textoOpcao == "-m") {
        quantMCs = atoi(argv[i + 1]);
      }
    }
  }

  hipSetDevice(idDevice);

  system((EXCLUIR_PASTA + string("Saidas")).c_str());

  string entrada, saida;
  for (int idMC = 0; idMC < quantMCs; idMC++) {
    entrada = string("Entradas");
    entrada += SEP;
    entrada += string("MonteCarlo_");
    entrada += to_string(idMC);
    entrada += SEP;

    saida = string("Saidas");
    saida += SEP;
    saida += string("MonteCarlo_");
    saida += to_string(idMC);
    saida += SEP;

    MonteCarlo(entrada, saida);
  }

  return 0;
}
