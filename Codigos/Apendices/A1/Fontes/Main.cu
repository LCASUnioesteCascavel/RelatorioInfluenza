#include <iostream>
#include <string>

#include "Fontes/Macros/MacrosSO.h"
#include "Fontes/MonteCarlo.h"

using std::cout;
using std::endl;
using std::string;
using std::to_string;

/*
  Metodo responsavel por apresentar uma tela de ajuda com as opcoes 
  disponiveis por linha de comando. Estas posicoes sao: 

  "--help -h": mostra a tela de ajuda; 
  "--device -d": especifica a GPU para execucao da simulacao; 
  "--nmontecarlos -m": especifica a quantidade de simulacoes Monte Carlo.
*/
void help(string exe) {
  cout << "Uso: " << exe << " <opcoes>\n"
       << "Opcoes:\n"
       << "\t--help, -h\t\tMostra esta ajuda\n"
       << "\t--device, -d ID_DEVICE\tEspecifica a GPU para execucao\n"
       << "\t--nmontecarlos, -m QUANT_MCS\tEspecifica a quantidade de simulacoes MC\n";
}

/*
  Metodo inicial do programa. 

  Este metodo e responsavel por:
  - interpretar as opcoes passadas por linha de comando;
  - alterar a GPU utilizada para execucao;
  - excluir pasta de saida se ja existente;
  - iniciar a execucao das simulacoes Monte Carlo. 

  Valores padrao:
  - "idDevice": 0. Utilizara a primeira GPU encontrada. A id das GPUs pode 
                   ser vista com o utilitario "deviceQuery" presente no 
                   SDK do CUDA. 
  - "quantMCS": 1. Executara uma simulacao Monte Carlo com os arquivos de 
                   entrada presentes na pasta "Entradas/MonteCarlo_0". 
                   Se informado um numero maior que 1 serao executadas 
                   simulacoes utilizando os arquivos presentes nas pastas 
                   "Entradas/MonteCarlo_{1}", em que "{1}" designa o id da 
                   simulacao, iniciando em "0" ate "quantMCS - 1".
*/
int main(int argc, char **argv) {
  int idDevice = 0, quantMCs = 1;

  // Interpreta os argumentos passados por linha do comando, se existentes. 
  if (argc > 1) {
    for (int i = 1; i < argc; i += 2) {
      string textoOpcao(argv[i]);

      if (textoOpcao == "--help" or textoOpcao == "-h") {
        help(argv[0]);
        return 0;
      }
      if (textoOpcao == "--device" or textoOpcao == "-d") {
        idDevice = atoi(argv[i + 1]);
      } 
      if (textoOpcao == "--nmontecarlos" or textoOpcao == "-m") {
        quantMCs = atoi(argv[i + 1]);
      }
    }
  }

  // Altera a GPU que sera utilizada para execucao. 
  hipSetDevice(idDevice);

  // Exclui a pasta de saida, se ja existente. 
  system((EXCLUIR_PASTA + string("Saidas")).c_str());

  string entrada, saida;
  for (int idMC = 0; idMC < quantMCs; idMC++) {
    // O caminho para a pasta de entrada sera "Entradas/MonteCarlo_{idMC}/"
    entrada = string("Entradas");
    entrada += SEP;
    entrada += string("MonteCarlo_");
    entrada += to_string(idMC);
    entrada += SEP;

    // O caminho para a pasta de saida sera "Saidas/MonteCarlo_{idMC}/"
    saida = string("Saidas");
    saida += SEP;
    saida += string("MonteCarlo_");
    saida += to_string(idMC);
    saida += SEP;

    // Inicia a execucao da simulacao tipo Monte Carlo. 
    MonteCarlo(entrada, saida);
  }

  return 0;
}
