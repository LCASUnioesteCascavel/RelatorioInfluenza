#include "hip/hip_runtime.h"
#include "Seeds.h"
#include "Fontes/Uteis/RandPerc.h"
#include "Fontes/Macros/MacrosParametros.h"

/*
  Operator () da classe InitSeeds.

  Este metodo e responsavel por inicializar as seeds para geracao de 
  numeros aleatorios em GPU. O primeiro elemento da tupla "t" e um id numerico 
  da seed e o segundo elemento e a estrutura utilizada para geracao dos 
  numeros aleatorios. 
*/
__host__ __device__ 
void InitSeeds::operator()(thrust::tuple<int, dre &> t) {
  int seed = get<0>(t);
  get<1>(t) = dre(seed);
}

/*
  Construtor da classe Seeds.

  A variavel "ind1" armazena uma instancia da classe "counting_iterator", que 
  mimetiza um contador infinito, comecando em "0". Esta variavel e utilizada a  
  geracao de ids a execucao dos metodos em GPU. Cada id indexa um dado que 
  pode ser processado paralelamente aos outros dados do conjunto. 

  O metodo realiza a geracao de numeros aleatorios em CPU e os utiliza a 
  geracao das seeds em GPU. 

  A quantidade de seeds geradas e obtida a partir do valor maximo encontrado 
  na lista "l". A lista "l" e inicializada com os valores das quantidades de 
  agentes humanos e posicoes do ambiente. 
*/
Seeds::Seeds(initializer_list<int> l) {
  // Inicializacao do contador em 0. 
  ind1 = make_counting_iterator(0);

  // Obtencao da quantidade de seeds necessarias. 
  nSeeds = max(l);

  RandPerc rand;

  // Geracao de numeros aleatorios em CPU.
  seedsDev = new DVector<dre>(nSeeds);
  int *rands = new int[nSeeds]();
  for (int i = 0; i < nSeeds; ++i) {
    rands[i] = (int)ENTRE_FAIXA(0, 100000, rand());
  }
  DVector<int> randsDev(rands, rands + nSeeds);
  delete[](rands);

  // Inicializacao das seeds em GPU. 
  for_each_n(
    make_zip_iterator(
      make_tuple(
        randsDev.begin(), seedsDev->begin()
      )
    ), nSeeds, InitSeeds()
  );
  PseedsDev = raw_pointer_cast(seedsDev->data());
}

/*
  Metodo responsavel pela obtencao do consumo de memiria da classe Seeds. 
*/
int Seeds::getMemoriaGPU() {
  return (nSeeds * sizeof(dre));
}

/*
  Destrutor da classe Seeds
*/
Seeds::~Seeds() {
  delete(seedsDev);
}
