#include "hip/hip_runtime.h"
#include "Seeds.h"
#include "Fontes/Uteis/RandPerc.h"
#include "Fontes/Macros/MacrosParametros.h"

__host__ __device__ 
void InitSeeds::operator()(tuple<int, dre &> t) {
  int seed = get<0>(t);
  get<1>(t) = dre(seed);
}

Seeds::Seeds(initializer_list<int> l) {
  ind1 = make_counting_iterator(0);

  nSeeds = max(l);

  RandPerc rand;

  seedsDev = new DVector<dre>(nSeeds);
  int *rands = new int[nSeeds]();
  for (int i = 0; i < nSeeds; ++i) {
    rands[i] = (int)ENTRE_FAIXA(0, 100000, rand());
  }
  DVector<int> randsDev(rands, rands + nSeeds);
  delete[](rands);

  for_each_n(
    make_zip_iterator(
      make_tuple(
        randsDev.begin(), seedsDev->begin()
      )
    ), nSeeds, InitSeeds()
  );
  PseedsDev = raw_pointer_cast(seedsDev->data());
}

int Seeds::getMemoriaGPU() {
  return (nSeeds * sizeof(dre));
}

Seeds::~Seeds() {
  delete(seedsDev);
}
