#include "Timer.h"

/*
  Construtor da classe Timer. 
*/
Timer::Timer() {
  this->total = 0;
}

/*
  Metodo responsavel por armazenar o tempo inicial da ocorrencia de um evento. 
*/
void Timer::start() {
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord(begin);
}

/*
  Metodo responsavel por armazenar o tempo final da ocorrencia de um evento. 
  Com os tempos iniciais e finais e possivel calcular o tempo dispendido em 
  uma operacao. 
*/
void Timer::stop() {
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&parcial, begin, end);
  total += parcial;
}

/*
  Retorna o tempo calculado em segundos. 
*/
double Timer::getTime() {
  return total / 1000;
}
