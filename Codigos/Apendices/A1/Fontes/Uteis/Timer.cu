#include "Timer.h"

Timer::Timer() {
  this->total = 0;
}

void Timer::start() {
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord(begin);
}

void Timer::stop() {
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&parcial, begin, end);
  total += parcial;
}

double Timer::getTime() {
  return total / 1000;
}
