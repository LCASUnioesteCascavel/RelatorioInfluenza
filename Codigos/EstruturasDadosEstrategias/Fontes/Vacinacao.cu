#include "hip/hip_runtime.h"
__host__ __device__ 
void Vacinacao::operator()(int id) {
  if (not periodoVacinacao()) return;

  dre seed = seeds[id];
  urd<double> dist(0.0, 1.0);

  int qVac = id;

  int fe_h, sd_h;
  int n[N_IDADES] = {0, 0, 0, 0};

  for (int idHumano = indHumanos[qVac]; 
       idHumano < indHumanos[qVac + 1]; ++idHumano) {
    fe_h = GET_FE_H(idHumano); sd_h = GET_SD_H(idHumano);

    if (sd_h == SUSCETIVEL) {
      n[fe_h]++;
    }
  }

  double percentualVacinacao = 1.0 / perVac[0];
  for (int fe = 0; fe < N_IDADES; ++fe) {
    if (faixaEtariaTeraVacinacao(fe)) {
      n[fe] = lround(n[fe] * percentualVacinacao);
    } else {
      n[fe] = 0;
    }
  }

  for (int idHumano = indHumanos[qVac]; 
       idHumano < indHumanos[qVac + 1]; ++idHumano) {
    fe_h = GET_FE_H(idHumano);
    sd_h = GET_SD_H(idHumano);

    if (sd_h == SUSCETIVEL and n[fe_h] > 0) {
      n[fe_h]--;

      if (randPerc <= TAXA_EFICACIA_VACINA) {
        SET_SD_H(idHumano, IMUNIZADO);
      }
    }
  }
}

__host__ __device__ 
void PosVacinacao::operator()(int id) {
  bool houveVacinacao = false;
  if (perVac[1] < perVac[0]) {
    for (int i = 0; i < sizeCicVac; ++i) {
      if (ciclo >= cicVac[i] and ciclo < (cicVac[i] + perVac[0])) {
        houveVacinacao = true;
        break;
      }
    }
  }
  if (houveVacinacao) perVac[1]++;
  else perVac[1] = 0;
}
