#include "hip/hip_runtime.h"
__host__ __device__ 
void Vacinacao::operator()(int id) {
  if (not periodoVacinacao()) return;

  dre seed = seeds[id];
  urd<double> dist(0.0, 1.0);

  // A vacinacao e aplicada em todas as quadras do ambiente. 
  int qVac = id;

  int fe_h, sd_h;
  int n[N_IDADES] = {0, 0, 0, 0};

  // Contagem, por faixas etarias, dos agentes presentes nesta quadra. 
  for (int idHumano = indHumanos[qVac]; 
       idHumano < indHumanos[qVac + 1]; ++idHumano) {
    fe_h = GET_FE_H(idHumano); sd_h = GET_SD_H(idHumano);

    if (sd_h == SUSCETIVEL) {
      n[fe_h]++;
    }
  }

  // Calcula as quantidades por faixas etarias de agentes que serao vacinados. 
  double percentualVacinacao = 1.0 / perVac[0];
  for (int fe = 0; fe < N_IDADES; ++fe) {
    if (faixaEtariaTeraVacinacao(fe)) {
      n[fe] = lround(n[fe] * percentualVacinacao);
    } else {
      n[fe] = 0;
    }
  }

  // Realiza a vacinacao para a quantidade de agentes calculada. 
  for (int idHumano = indHumanos[qVac]; 
       idHumano < indHumanos[qVac + 1]; ++idHumano) {
    fe_h = GET_FE_H(idHumano);
    sd_h = GET_SD_H(idHumano);

    if (sd_h == SUSCETIVEL and n[fe_h] > 0) {
      n[fe_h]--;

      // O agente pode ser passado probabilisticamente para o estado imunizado. 
      if (randPerc <= TAXA_EFICACIA_VACINA) {
        SET_SD_H(idHumano, IMUNIZADO);
      }
    }
  }
}

PosVacinacao::PosVacinacao(Ambiente *ambiente, int ciclo, 
                           int sizePerVac, int sizeCicVac) {
  this->ciclo = ciclo;
  this->sizePerVac = sizePerVac;
  this->sizeCicVac = sizeCicVac;
  this->perVac = ambiente->PperVacDev;
  this->cicVac = ambiente->PcicVacDev;
}
